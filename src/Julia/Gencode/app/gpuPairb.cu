#include "hip/hip_runtime.h"
template <typename T>  __global__  void kernelgpuPairb1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu1 = mu[0];
		T mu2 = mu[1];
		T mu3 = mu[2];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		f[0 + i*1] = -mu1*exp(2*mu2*(mu3 - sqrt(pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2))));
		i *= blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPairb1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPairb1<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __global__  void kernelgpuPairb2(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu4 = mu[3];
		T mu5 = mu[4];
		T mu6 = mu[5];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = -mu5;
		f[0 + i*1] = mu4*(-pow(mu6 + x0, 2) + pow(x0 + sqrt(pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2)), 2));
		i *= blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPairb2(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPairb2<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T> void gpuPairb(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPairb1(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 2)
		gpuPairb2(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPairb(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairb(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
