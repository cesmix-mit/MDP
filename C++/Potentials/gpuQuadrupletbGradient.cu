
#include <hip/hip_runtime.h>
template <typename T> void gpuQuadrupletbGradient(T *u, T *u_xij, T *u_xik, T *u_xil, T *xij, T *xik, T *xil, T *qi, T *qj, T *qk, T *ql, int *ti, int *tj, int *tk, int *tl, int *ai, int *aj, int *ak, int *al, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuQuadrupletbGradient(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, int *, int *, int *, int *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuQuadrupletbGradient(float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, int *, int *, int *, int *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
