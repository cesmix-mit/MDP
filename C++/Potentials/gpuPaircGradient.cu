
#include <hip/hip_runtime.h>
template <typename T> void gpuPaircGradient(T *u, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPaircGradient(double *, double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPaircGradient(float *, float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuPaircDensityGradient(T *u, T *u_rho, T *rho, T *mu, T *eta, int *kappa, int nrho, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPaircDensityGradient(double *, double *, double *, double *, double *, int*, int, int, int, int, int, int);
template void gpuPaircDensityGradient(float *, float *, float *, float *, float *, int *, int, int, int, int, int, int);

