#include "hip/hip_runtime.h"
template <typename T>  __global__  void kernelgpuPairb1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu1 = mu[0];
		T mu2 = mu[1];
		T mu3 = mu[2];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		u[i] = -mu1*exp(2*mu2*(mu3 - sqrt(pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2))));
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPairb1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPairb1<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __device__  void devicegpuPairb1(T *__restrict__ u, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu1 = mu[0];
		T mu2 = mu[1];
		T mu3 = mu[2];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		u[i] = -mu1*exp(2*mu2*(mu3 - sqrt(pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2))));
		i += blockDim.x * gridDim.x;
	}
}


template <typename T>  __global__  void kernelgpuPairb1Gradient(T *__restrict__ u, T *__restrict__ du, T *__restrict__ u_xij, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	__enzyme_autodiff((void*)devicegpuPairb1<T>, 
		enzyme_dup, u, du, 
		enzyme_dup, xij, u_xij, 
		enzyme_const, qi, 
		enzyme_const, qj, 
		enzyme_const, ti, 
		enzyme_const, tj, 
		enzyme_const, ai, 
		enzyme_const, aj, 
		enzyme_const, mu, 
		enzyme_const, eta, 
		enzyme_const, kappa, 
		dim, ncq, nmu, neta, nkappa, ng); 
}

template <typename T> void gpuPairb1Gradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPairb1Gradient<<<gridDim, blockDim>>>(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __global__  void kernelgpuPairb2(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu4 = mu[3];
		T mu5 = mu[4];
		T mu6 = mu[5];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = -mu5;
		u[i] = mu4*(-pow(mu6 + x0, 2) + pow(x0 + sqrt(pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2)), 2));
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPairb2(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPairb2<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __device__  void devicegpuPairb2(T *__restrict__ u, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu4 = mu[3];
		T mu5 = mu[4];
		T mu6 = mu[5];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = -mu5;
		u[i] = mu4*(-pow(mu6 + x0, 2) + pow(x0 + sqrt(pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2)), 2));
		i += blockDim.x * gridDim.x;
	}
}


template <typename T>  __global__  void kernelgpuPairb2Gradient(T *__restrict__ u, T *__restrict__ du, T *__restrict__ u_xij, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	__enzyme_autodiff((void*)devicegpuPairb2<T>, 
		enzyme_dup, u, du, 
		enzyme_dup, xij, u_xij, 
		enzyme_const, qi, 
		enzyme_const, qj, 
		enzyme_const, ti, 
		enzyme_const, tj, 
		enzyme_const, ai, 
		enzyme_const, aj, 
		enzyme_const, mu, 
		enzyme_const, eta, 
		enzyme_const, kappa, 
		dim, ncq, nmu, neta, nkappa, ng); 
}

template <typename T> void gpuPairb2Gradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPairb2Gradient<<<gridDim, blockDim>>>(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T> void gpuPairb(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPairb1(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 2)
		gpuPairb2(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPairb(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairb(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuPairbGradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPairb1Gradient(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 2)
		gpuPairb2Gradient(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPairbGradient(double *, double *, double*, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairbGradient(float *, float *, float*, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
