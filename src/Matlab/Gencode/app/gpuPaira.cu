
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuPaira1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu1 = mu[0];
		T mu2 = mu[1];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T t2 = xij1*xij1;
		T t3 = xij2*xij2;
		T t4 = xij3*xij3;
		T t5 = t2+t3+t4;
		u[i] = -mu2*1.0/(t5*t5*t5)+mu1*1.0/(t5*t5*t5*t5*t5*t5);
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPaira1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira1<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T> void gpuPaira(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPaira1(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPaira(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPaira(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
