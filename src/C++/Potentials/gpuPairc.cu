
#include <hip/hip_runtime.h>
template <typename T> void gpuPairc(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPairc(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairc(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuPaircDensity(T *u, T *rho, T *mu, T *eta, int *kappa, int nrho, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPaircDensity(double *, double *, double *, double *, int*, int, int, int, int, int, int);
template void gpuPaircDensity(float *, float *, float *, float *, int *, int, int, int, int, int, int);

