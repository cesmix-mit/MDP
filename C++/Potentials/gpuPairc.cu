
#include <hip/hip_runtime.h>
template <typename T> void gpuPairc(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPairc(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairc(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuPaircGradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPaircGradient(double *, double *, double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPaircGradient(float *, float *, float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuPaircDensity(T *u, T *rho, T *mu, T *eta, int *kappa, int nrho, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPaircDensity(double *, double *, double *, double *, int*, int, int, int, int, int, int);
template void gpuPaircDensity(float *, float *, float *, float *, int *, int, int, int, int, int, int);

template <typename T> void gpuPaircDensityGradient(T *u, T *du, T *u_rho, T *rho, T *mu, T *eta, int *kappa, int nrho, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPaircDensityGradient(double *, double *, double *, double *, double *, double *, int*, int, int, int, int, int, int);
template void gpuPaircDensityGradient(float *, float *, float *, float *, float *, float *, int *, int, int, int, int, int, int);

