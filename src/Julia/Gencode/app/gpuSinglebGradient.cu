
#include <hip/hip_runtime.h>
template <typename T> void gpuSinglebGradient(T *u, T *u_xi, T *xi, T *qi, int *ti, int *ai, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuSinglebGradient(double *, double *, double *, double *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuSinglebGradient(float *, float *, float *, float *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
