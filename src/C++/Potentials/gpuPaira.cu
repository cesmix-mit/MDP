
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuPaira1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		int ti1 = ti[0 + i*1];
		int tj1 = tj[0 + i*1];
		T t2 = xij1*xij1;
		T t3 = xij2*xij2;
		T t4 = xij3*xij3;
		T t5 = t2+t3+t4;
		T t6 = sqrt(t5);
		T t7 = t6-4.0;
		T t8 = t7*t7;
		T t9 = ti1*3.4E1;
		T t10 = tj1*3.4E1;
		T t11 = -t9+8.3E1;
		T t12 = pow(t11,2.3E1/1.0E2);
		T t13 = -t10+8.3E1;
		T t14 = pow(t13,2.3E1/1.0E2);
		T t29 = t12*2.868550693703308E1;
		T t30 = t14*2.868550693703308E1;
		T t15 = -t29-t30;
		T t16 = exp(t15);
		T t26 = t12*1.807479188900747;
		T t27 = t14*1.807479188900747;
		T t17 = -t26-t27;
		T t18 = exp(t17);
		T t23 = t12*3.611910352187834;
		T t24 = t14*3.611910352187834;
		T t19 = -t23-t24;
		T t20 = exp(t19);
		T t32 = t12*8.447423692636073;
		T t33 = t14*8.447423692636073;
		T t21 = -t32-t33;
		T t22 = exp(t21);
		T t57 = t12*8.599786552828175E-1;
		T t58 = t14*8.599786552828175E-1;
		T t25 = t57+t58;
		T t61 = t12*4.303521878335112E-1;
		T t62 = t14*4.303521878335112E-1;
		T t28 = t61+t62;
		T t65 = t12*6.829882604055496;
		T t66 = t14*6.829882604055496;
		T t31 = t65+t66;
		T t69 = t12*2.011291355389541;
		T t70 = t14*2.011291355389541;
		T t34 = t69+t70;
		T t35 = t9-8.3E1;
		T t36 = t10-8.3E1;
		T t37 = t12*1.241331163287086;
		T t38 = t14*1.241331163287086;
		T t39 = t37+t38;
		T t40 = t12*1.212302113127001E-2;
		T t41 = t14*1.212302113127001E-2;
		T t42 = t40+t41;
		T t43 = t12*2.409832187833511E-1;
		T t44 = t14*2.409832187833511E-1;
		T t45 = t43+t44;
		T t46 = t12*1.025477010458911;
		T t47 = t14*1.025477010458911;
		T t48 = t46+t47;
		T t49 = t20*3.177097505668934E-2;
		T t50 = t18*3.193877551020408E-3;
		T t51 = t16*2.060657596371882E-2;
		T t52 = t22*5.780725623582766E-2;
		T t53 = t16*t39*(1.0E1/2.1E1);
		T t54 = t18*t42*(1.0E1/2.1E1);
		T t55 = t20*t45*(1.0E1/2.1E1);
		T t56 = t22*t48*(1.0E1/2.1E1);
		T t59 = t25*t25;
		T t60 = t20*t59*2.8022E-1;
		T t63 = t28*t28;
		T t64 = t18*t63*2.817E-2;
		T t67 = t31*t31;
		T t68 = t16*t67*1.8175E-1;
		T t71 = t34*t34;
		T t72 = t22*t71*5.0986E-1;
		T t73 = t49+t50+t51+t52+t53+t54+t55+t56+t60+t64+t68+t72;
		T t74 = t20*6.671904761904762E-2;
		T t75 = t18*6.707142857142857E-3;
		T t76 = t16*4.327380952380952E-2;
		T t77 = t22*1.213952380952381E-1;
		T t78 = t16*t39;
		T t79 = t18*t42;
		T t80 = t20*t45;
		T t81 = t22*t48;
		T t82 = t74+t75+t76+t77+t78+t79+t80+t81;
		T t83 = t6*1.0E2;
		T t84 = t83-4.0E2;
		T t85 = tanh(t84);
		T t86 = t85*(1.0/2.0);
		T t87 = t20*2.8022E-1;
		T t88 = t18*2.817E-2;
		T t89 = t16*1.8175E-1;
		T t90 = t22*5.0986E-1;
		T t91 = t87+t88+t89+t90;
		T t92 = t35*t36*t91*3.428486904761905;
		T t93 = t35*t36*t73*1.142828968253968E-2;
		T t94 = t35*t36*t82*3.428486904761905E-1;
		u[i] = -(t86+1.0/2.0)*(t92+t93+t94-t7*t8*(-t7*(t35*t36*t73*2.14280431547619E1+t35*t36*t82*2.14280431547619E2)+t35*t36*t73*5.714144841269841+t35*t36*t82*8.571217261904762E1))+(t86-1.0/2.0)*(t92+t93+t94)+1.0/sqrt(t5)*t35*t36*(exp(-t6*t25)*2.8022E-1+exp(-t6*t28)*2.817E-2+exp(-t6*t31)*1.8175E-1+exp(-t6*t34)*5.0986E-1)*1.4399645E1;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPaira1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira1<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

#ifdef _ENZYME
template <typename T>  __device__  void devicegpuPaira1(T *__restrict__ u, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		int ti1 = ti[0 + i*1];
		int tj1 = tj[0 + i*1];
		T t2 = xij1*xij1;
		T t3 = xij2*xij2;
		T t4 = xij3*xij3;
		T t5 = t2+t3+t4;
		T t6 = sqrt(t5);
		T t7 = t6-4.0;
		T t8 = t7*t7;
		T t9 = ti1*3.4E1;
		T t10 = tj1*3.4E1;
		T t11 = -t9+8.3E1;
		T t12 = pow(t11,2.3E1/1.0E2);
		T t13 = -t10+8.3E1;
		T t14 = pow(t13,2.3E1/1.0E2);
		T t29 = t12*2.868550693703308E1;
		T t30 = t14*2.868550693703308E1;
		T t15 = -t29-t30;
		T t16 = exp(t15);
		T t26 = t12*1.807479188900747;
		T t27 = t14*1.807479188900747;
		T t17 = -t26-t27;
		T t18 = exp(t17);
		T t23 = t12*3.611910352187834;
		T t24 = t14*3.611910352187834;
		T t19 = -t23-t24;
		T t20 = exp(t19);
		T t32 = t12*8.447423692636073;
		T t33 = t14*8.447423692636073;
		T t21 = -t32-t33;
		T t22 = exp(t21);
		T t57 = t12*8.599786552828175E-1;
		T t58 = t14*8.599786552828175E-1;
		T t25 = t57+t58;
		T t61 = t12*4.303521878335112E-1;
		T t62 = t14*4.303521878335112E-1;
		T t28 = t61+t62;
		T t65 = t12*6.829882604055496;
		T t66 = t14*6.829882604055496;
		T t31 = t65+t66;
		T t69 = t12*2.011291355389541;
		T t70 = t14*2.011291355389541;
		T t34 = t69+t70;
		T t35 = t9-8.3E1;
		T t36 = t10-8.3E1;
		T t37 = t12*1.241331163287086;
		T t38 = t14*1.241331163287086;
		T t39 = t37+t38;
		T t40 = t12*1.212302113127001E-2;
		T t41 = t14*1.212302113127001E-2;
		T t42 = t40+t41;
		T t43 = t12*2.409832187833511E-1;
		T t44 = t14*2.409832187833511E-1;
		T t45 = t43+t44;
		T t46 = t12*1.025477010458911;
		T t47 = t14*1.025477010458911;
		T t48 = t46+t47;
		T t49 = t20*3.177097505668934E-2;
		T t50 = t18*3.193877551020408E-3;
		T t51 = t16*2.060657596371882E-2;
		T t52 = t22*5.780725623582766E-2;
		T t53 = t16*t39*(1.0E1/2.1E1);
		T t54 = t18*t42*(1.0E1/2.1E1);
		T t55 = t20*t45*(1.0E1/2.1E1);
		T t56 = t22*t48*(1.0E1/2.1E1);
		T t59 = t25*t25;
		T t60 = t20*t59*2.8022E-1;
		T t63 = t28*t28;
		T t64 = t18*t63*2.817E-2;
		T t67 = t31*t31;
		T t68 = t16*t67*1.8175E-1;
		T t71 = t34*t34;
		T t72 = t22*t71*5.0986E-1;
		T t73 = t49+t50+t51+t52+t53+t54+t55+t56+t60+t64+t68+t72;
		T t74 = t20*6.671904761904762E-2;
		T t75 = t18*6.707142857142857E-3;
		T t76 = t16*4.327380952380952E-2;
		T t77 = t22*1.213952380952381E-1;
		T t78 = t16*t39;
		T t79 = t18*t42;
		T t80 = t20*t45;
		T t81 = t22*t48;
		T t82 = t74+t75+t76+t77+t78+t79+t80+t81;
		T t83 = t6*1.0E2;
		T t84 = t83-4.0E2;
		T t85 = tanh(t84);
		T t86 = t85*(1.0/2.0);
		T t87 = t20*2.8022E-1;
		T t88 = t18*2.817E-2;
		T t89 = t16*1.8175E-1;
		T t90 = t22*5.0986E-1;
		T t91 = t87+t88+t89+t90;
		T t92 = t35*t36*t91*3.428486904761905;
		T t93 = t35*t36*t73*1.142828968253968E-2;
		T t94 = t35*t36*t82*3.428486904761905E-1;
		u[i] = -(t86+1.0/2.0)*(t92+t93+t94-t7*t8*(-t7*(t35*t36*t73*2.14280431547619E1+t35*t36*t82*2.14280431547619E2)+t35*t36*t73*5.714144841269841+t35*t36*t82*8.571217261904762E1))+(t86-1.0/2.0)*(t92+t93+t94)+1.0/sqrt(t5)*t35*t36*(exp(-t6*t25)*2.8022E-1+exp(-t6*t28)*2.817E-2+exp(-t6*t31)*1.8175E-1+exp(-t6*t34)*5.0986E-1)*1.4399645E1;
		i += blockDim.x * gridDim.x;
	}
}


template <typename T>  __global__  void kernelgpuPaira1Gradient(T *__restrict__ u, T *__restrict__ du, T *__restrict__ u_xij, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	__enzyme_autodiff((void*)devicegpuPaira1<T>, 
		enzyme_dup, u, du, 
		enzyme_dup, xij, u_xij, 
		enzyme_const, qi, 
		enzyme_const, qj, 
		enzyme_const, ti, 
		enzyme_const, tj, 
		enzyme_const, ai, 
		enzyme_const, aj, 
		enzyme_const, mu, 
		enzyme_const, eta, 
		enzyme_const, kappa, 
		dim, ncq, nmu, neta, nkappa, ng); 
}

template <typename T> void gpuPaira1Gradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira1Gradient<<<gridDim, blockDim>>>(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
#endif

template <typename T> void gpuPaira(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPaira1(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPaira(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPaira(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

#ifdef _ENZYME
template <typename T> void gpuPairaGradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPaira1Gradient(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPairaGradient(double *, double *, double*, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairaGradient(float *, float *, float*, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
#endif
