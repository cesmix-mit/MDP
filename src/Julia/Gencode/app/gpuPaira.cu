#include "hip/hip_runtime.h"
template <typename T>  __global__  void kernelgpuPaira1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu1 = mu[0];
		T mu2 = mu[1];
		T eta1 = eta[0];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2);
		u[i] = mu1/pow(x0, 6) - mu2/pow(x0, 3) + mu2/pow(eta1, 6) - mu1/pow(eta1, 12);
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPaira1(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira1<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __device__  void devicegpuPaira1(T *__restrict__ u, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu1 = mu[0];
		T mu2 = mu[1];
		T eta1 = eta[0];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2);
		u[i] = mu1/pow(x0, 6) - mu2/pow(x0, 3) + mu2/pow(eta1, 6) - mu1/pow(eta1, 12);
		i += blockDim.x * gridDim.x;
	}
}


template <typename T>  __global__  void kernelgpuPaira1Gradient(T *__restrict__ u, T *__restrict__ du, T *__restrict__ u_xij, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	__enzyme_autodiff((void*)devicegpuPaira1<T>, 
		enzyme_dup, u, du, 
		enzyme_dup, xij, u_xij, 
		enzyme_const, qi, 
		enzyme_const, qj, 
		enzyme_const, ti, 
		enzyme_const, tj, 
		enzyme_const, ai, 
		enzyme_const, aj, 
		enzyme_const, mu, 
		enzyme_const, eta, 
		enzyme_const, kappa, 
		dim, ncq, nmu, neta, nkappa, ng); 
}

template <typename T> void gpuPaira1Gradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira1Gradient<<<gridDim, blockDim>>>(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __global__  void kernelgpuPaira2(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu3 = mu[2];
		T mu4 = mu[3];
		T eta2 = eta[1];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T qi1 = qi[0 + i*1];
		T qj1 = qj[0 + i*1];
		T x0 = pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2);
		u[i] = mu4*qi1*qj1*pow(pow(x0, 3.0/2.0) + pow(mu3, -3), -0.33333333333333331)*(1 - 35*pow(x0, 2)/pow(eta2, 4) + 84*pow(x0, 5.0/2.0)/pow(eta2, 5) - 70*pow(x0, 3)/pow(eta2, 6) + 20*pow(x0, 7.0/2.0)/pow(eta2, 7));
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPaira2(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira2<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __device__  void devicegpuPaira2(T *__restrict__ u, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu3 = mu[2];
		T mu4 = mu[3];
		T eta2 = eta[1];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T qi1 = qi[0 + i*1];
		T qj1 = qj[0 + i*1];
		T x0 = pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2);
		u[i] = mu4*qi1*qj1*pow(pow(x0, 3.0/2.0) + pow(mu3, -3), -0.33333333333333331)*(1 - 35*pow(x0, 2)/pow(eta2, 4) + 84*pow(x0, 5.0/2.0)/pow(eta2, 5) - 70*pow(x0, 3)/pow(eta2, 6) + 20*pow(x0, 7.0/2.0)/pow(eta2, 7));
		i += blockDim.x * gridDim.x;
	}
}


template <typename T>  __global__  void kernelgpuPaira2Gradient(T *__restrict__ u, T *__restrict__ du, T *__restrict__ u_xij, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	__enzyme_autodiff((void*)devicegpuPaira2<T>, 
		enzyme_dup, u, du, 
		enzyme_dup, xij, u_xij, 
		enzyme_const, qi, 
		enzyme_const, qj, 
		enzyme_const, ti, 
		enzyme_const, tj, 
		enzyme_const, ai, 
		enzyme_const, aj, 
		enzyme_const, mu, 
		enzyme_const, eta, 
		enzyme_const, kappa, 
		dim, ncq, nmu, neta, nkappa, ng); 
}

template <typename T> void gpuPaira2Gradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira2Gradient<<<gridDim, blockDim>>>(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __global__  void kernelgpuPaira3(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu5 = mu[4];
		T mu6 = mu[5];
		T mu7 = mu[6];
		T mu8 = mu[7];
		T mu9 = mu[8];
		T eta3 = eta[2];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2);
		T x1 = mu7*(1 - pow(pow(x0, (1.0/2.0)*mu9) + pow(1.0/mu5, mu9), 1.0/mu9)/mu8);
		u[i] = mu6*(-2*exp(0.5*x1) + exp(x1))*(1 - 35*pow(x0, 2)/pow(eta3, 4) + 84*pow(x0, 5.0/2.0)/pow(eta3, 5) - 70*pow(x0, 3)/pow(eta3, 6) + 20*pow(x0, 7.0/2.0)/pow(eta3, 7));
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuPaira3(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira3<<<gridDim, blockDim>>>(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T>  __device__  void devicegpuPaira3(T *__restrict__ u, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int i = threadIdx.x * blockIdx.x * blockDim.x;
	while (i<ng) {
		T mu5 = mu[4];
		T mu6 = mu[5];
		T mu7 = mu[6];
		T mu8 = mu[7];
		T mu9 = mu[8];
		T eta3 = eta[2];
		T xij1 = xij[0 + i*3];
		T xij2 = xij[1 + i*3];
		T xij3 = xij[2 + i*3];
		T x0 = pow(xij1, 2) + pow(xij2, 2) + pow(xij3, 2);
		T x1 = mu7*(1 - pow(pow(x0, (1.0/2.0)*mu9) + pow(1.0/mu5, mu9), 1.0/mu9)/mu8);
		u[i] = mu6*(-2*exp(0.5*x1) + exp(x1))*(1 - 35*pow(x0, 2)/pow(eta3, 4) + 84*pow(x0, 5.0/2.0)/pow(eta3, 5) - 70*pow(x0, 3)/pow(eta3, 6) + 20*pow(x0, 7.0/2.0)/pow(eta3, 7));
		i += blockDim.x * gridDim.x;
	}
}


template <typename T>  __global__  void kernelgpuPaira3Gradient(T *__restrict__ u, T *__restrict__ du, T *__restrict__ u_xij, T *__restrict__ xij, T *__restrict__ qi, T *__restrict__ qj, int *__restrict__ ti, int *__restrict__ tj, int *__restrict__ ai, int *__restrict__ aj, T *__restrict__ mu, T *__restrict__ eta, int *__restrict__ kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	__enzyme_autodiff((void*)devicegpuPaira3<T>, 
		enzyme_dup, u, du, 
		enzyme_dup, xij, u_xij, 
		enzyme_const, qi, 
		enzyme_const, qj, 
		enzyme_const, ti, 
		enzyme_const, tj, 
		enzyme_const, ai, 
		enzyme_const, aj, 
		enzyme_const, mu, 
		enzyme_const, eta, 
		enzyme_const, kappa, 
		dim, ncq, nmu, neta, nkappa, ng); 
}

template <typename T> void gpuPaira3Gradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng)
{
	int blockDim = 256;
	int gridDim = (ng * blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuPaira3Gradient<<<gridDim, blockDim>>>(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}

template <typename T> void gpuPaira(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPaira1(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 2)
		gpuPaira2(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 3)
		gpuPaira3(u, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPaira(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPaira(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuPairaGradient(T *u, T *du, T *u_xij, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
	if (potnum == 1)
		gpuPaira1Gradient(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 2)
		gpuPaira2Gradient(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
	else if (potnum == 3)
		gpuPaira3Gradient(u, du, u_xij, xij, qi, qj, ti, tj, ai, aj, mu, eta, kappa, dim, ncq, nmu, neta, nkappa, ng);
}
template void gpuPairaGradient(double *, double *, double*, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairaGradient(float *, float *, float*, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
