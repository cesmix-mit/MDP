
#include <hip/hip_runtime.h>
template <typename T> void gpuQuadrupleta(T *u, T *xij, T *xik, T *xil, T *qi, T *qj, T *qk, T *ql, int *ti, int *tj, int *tk, int *tl, int *ai, int *aj, int *ak, int *al, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuQuadrupleta(double *, double *, double *, double *, double *, double *, double *, double *, int *, int *, int *, int *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuQuadrupleta(float *, float *, float *, float *, float *, float *, float *, float *, int *, int *, int *, int *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);

template <typename T> void gpuQuadrupletaGradient(T *u, T *du, T *u_xij, T *u_xik, T *u_xil, T *xij, T *xik, T *xil, T *qi, T *qj, T *qk, T *ql, int *ti, int *tj, int *tk, int *tl, int *ai, int *aj, int *ak, int *al, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuQuadrupletaGradient(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, int *, int *, int *, int *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuQuadrupletaGradient(float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, int *, int *, int *, int *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
