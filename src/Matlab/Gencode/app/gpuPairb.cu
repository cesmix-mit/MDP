
#include <hip/hip_runtime.h>
template <typename T> void gpuPairb(T *u, T *xij, T *qi, T *qj, int *ti, int *tj, int *ai, int *aj, T *mu, T *eta, int *kappa, int dim, int ncq, int nmu, int neta, int nkappa, int ng, int potnum)
{
}
template void gpuPairb(double *, double *, double *, double *, int *, int *, int *, int *, double *, double *, int*, int, int, int, int, int, int, int);
template void gpuPairb(float *, float *, float *, float *, int *, int *, int *, int *, float *, float *, int *, int, int, int, int, int, int, int);
