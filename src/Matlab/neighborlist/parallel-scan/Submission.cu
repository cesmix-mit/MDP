/*
	Matt Dean - 1422434 - mxd434
	
	Goals implemented:
		- Block scan for arbitrary length small vectors - 'blockscan' function
		- Full scan for arbitrary length large vectors	- 'scan' function
			This function decides whether to perform a small (one block) scan or a full (n-level) scan depending on the length of the input vector
		- BCAO for both scans

	Hardware:
		CPU - Intel Core i5-4670k @ 3.4GHz
		GPU - NVIDIA GeForce GTX 760

	Timings:
		10,000,000 Elements
		  host     : 20749 ms
		  gpu      : 7.860768 ms
		  gpu bcao : 4.304064 ms
		
		For more results please see the comment at the bottom of this file

	Extra work:
		Due to the recursive nature of the full scan it can handle n > 3 levels 
	
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"

#include "hip/device_functions.h"

// scan.cuh
long sequential_scan(int* output, int* input, int length);
float blockscan(int *output, int *input, int length, bool bcao);
float scan(int *output, int *input, int length, bool bcao);

void scanLargeDeviceArray(int *output, int *input, int length, bool bcao);
void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao);
void scanLargeEvenDeviceArray(int *output, int *input, int length, bool bcao);


// kernels.cuh
__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo);
__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo);

__global__ void prescan_large(int *output, int *input, int n, int* sums);
__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums);

__global__ void add(int *output, int length, int *n1);
__global__ void add(int *output, int length, int *n1, int *n2);


// utils.h
void _checkCudaError(const char *message, hipError_t err, const char *caller);
void printResult(const char* prefix, int result, long nanoseconds);
void printResult(const char* prefix, int result, float milliseconds);
void printiarray(int* a, int m);

bool isPowerOfTwo(int x);
int nextPowerOfTwo(int x);

long get_nanos();


__global__ void gpuKernelPrintArray(int * a, int m)
{        
    for (int i=0; i<m; i++) 
            printf("%i   ", a[i]);                
    printf("\n");
}

void gpuPrintArray(int * a, int m)
{
    gpuKernelPrintArray<<<1, 1>>>(a, m);
}

/*///////////////////////////////////*/
/*            Main.cpp               */
/*///////////////////////////////////*/
void test(int N) {
	bool canBeBlockscanned = N <= 1024;

	time_t t;
	srand((unsigned)time(&t));
	int *in = new int[N];
	for (int i = 0; i < N; i++) {
		in[i] = rand() % 10;
	}

	printf("%i Elements \n", N);

	// sequential scan on CPU
	int *outHost = new int[N]();
	long time_host = sequential_scan(outHost, in, N);
	printResult("host    ", outHost[N - 1], time_host);

	// full scan
	int *outGPU = new int[N]();
	float time_gpu = scan(outGPU, in, N, false);
	printResult("gpu     ", outGPU[N - 1], time_gpu);

	// full scan with BCAO
	int *outGPU_bcao = new int[N]();
	float time_gpu_bcao = scan(outGPU_bcao, in, N, true);
	printResult("gpu bcao", outGPU_bcao[N - 1], time_gpu_bcao);

    hipDeviceSynchronize();
    /*
    if (N<200) {
        printiarray(in, N);
        printiarray(outHost, N);
        printiarray(outGPU, N);
        printiarray(outGPU_bcao, N);
    }
`   */

	if (canBeBlockscanned) {
		// basic level 1 block scan
		int *out_1block = new int[N]();
		float time_1block = blockscan(out_1block, in, N, false);
		printResult("level 1 ", out_1block[N - 1], time_1block);
       
		// level 1 block scan with BCAO
		int *out_1block_bcao = new int[N]();
		float time_1block_bcao = blockscan(out_1block_bcao, in, N, true);
		printResult("l1 bcao ", out_1block_bcao[N - 1], time_1block_bcao);

        //cudaDeviceSynchronize();
        //printiarray(out_1block, N);
        //printiarray(out_1block_bcao, N);

		delete[] out_1block;
		delete[] out_1block_bcao;
	}

	printf("\n");

	delete[] in;
	delete[] outHost;
	delete[] outGPU;
	delete[] outGPU_bcao;
}

int main()
{
	int TEN_MILLION = 10000000;
	int ONE_MILLION = 1000000;
	int TEN_THOUSAND = 10000;

	int elements[] = {
		TEN_MILLION * 2,
		TEN_MILLION,
		ONE_MILLION,
		TEN_THOUSAND,
		5000,
		4096,
		2048,
		2000,
		1000,
		500,
		100,
		64,
		8,
		5
	};

	int numElements = sizeof(elements) / sizeof(elements[0]);

	for (int i = 0; i < numElements; i++) {
		test(elements[i]);
	}

	return 0;
}



/*///////////////////////////////////*/
/*            scan.cu                */
/*///////////////////////////////////*/
#define checkCudaError(o, l) _checkCudaError(o, l, __func__)

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

long sequential_scan(int* output, int* input, int length) {
	long start_time = get_nanos();

	output[0] = 0; // since this is a prescan, not a scan
	for (int j = 1; j < length; ++j)
	{
		output[j] = input[j - 1] + output[j - 1];
	}

	long end_time = get_nanos();
	return end_time - start_time;
}

float blockscan(int *output, int *input, int length, bool bcao) {
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int powerOfTwo = nextPowerOfTwo(length);
	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

    //printiarray(input, length);
    //printiarray(output, length);
    gpuPrintArray(d_in, length);
    gpuPrintArray(d_out, length);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

float scan(int *output, int *input, int length, bool bcao) {
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);


    //long start_time = get_nanos();
	if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		scanSmallDeviceArray(d_out, d_in, length, bcao);
	}
    //long end_time = get_nanos();

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
    //return end_time - start_time;
}


void scanLargeDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		// perform a large scan on a compatible multiple of elements
		int lengthMultiple = length - remainder;
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		int *startOfOutputArray = &(d_out[lengthMultiple]);
		scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

		add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	}
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int powerOfTwo = nextPowerOfTwo(length);

	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	int *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	if (bcao) {
		prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}
	else {
		prescan_large_unoptimized<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
		// perform a large scan on the sums arr
		scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
	}
	else {
		// only need one block to scan sums arr so can use small scan
		scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
	}

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

	hipFree(d_sums);
	hipFree(d_incr);
}



/*///////////////////////////////////*/
/*            kernels.cu             */
/*///////////////////////////////////*/
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5

// There were two BCAO optimisations in the paper - this one is fastest
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo)
{
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


	if (threadID < n) {
		temp[ai + bankOffsetA] = input[ai];
		temp[bi + bankOffsetB] = input[bi];
	}
	else {
		temp[ai + bankOffsetA] = 0;
		temp[bi + bankOffsetB] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) {
		temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
	}

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[ai] = temp[ai + bankOffsetA];
		output[bi] = temp[bi + bankOffsetB];
	}
}

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	if (threadID < n) {
		temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
		temp[2 * threadID + 1] = input[2 * threadID + 1];
	}
	else {
		temp[2 * threadID] = 0;
		temp[2 * threadID + 1] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[2 * threadID] = temp[2 * threadID]; // write results to device memory
		output[2 * threadID + 1] = temp[2 * threadID + 1];
	}
}


__global__ void prescan_large(int *output, int *input, int n, int *sums) {
	extern __shared__ int temp[];

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = input[blockOffset + ai];
	temp[bi + bankOffsetB] = input[blockOffset + bi];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + ai] = temp[ai + bankOffsetA];
	output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	extern __shared__ int temp[];
	temp[2 * threadID] = input[blockOffset + (2 * threadID)];
	temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1];
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + (2 * threadID)] = temp[2 * threadID];
	output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}


__global__ void add(int *output, int length, int *n) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}


/*///////////////////////////////////*/
/*            utils.cpp              */
/*///////////////////////////////////*/
void _checkCudaError(const char *message, hipError_t err, const char *caller) {
	if (err != hipSuccess) {
		fprintf(stderr, "Error in: %s\n", caller);
		fprintf(stderr, message);
		fprintf(stderr, ": %s\n", hipGetErrorString(err));
		exit(0);
	}
}

void printResult(const char* prefix, int result, long nanoseconds) {
	printf("  ");
	printf(prefix);
	printf(" : %i in %ld ms \n", result, nanoseconds / 1000);
}

void printResult(const char* prefix, int result, float milliseconds) {
	printf("  ");
	printf(prefix);
	printf(" : %i in %f ms \n", result, milliseconds);
}

void printiarray(int* a, int m)
{    
    for (int i=0; i<m; i++)
        printf("%i  ", a[i]);
    printf("\n");
}

// from https://stackoverflow.com/a/3638454
bool isPowerOfTwo(int x) {
	return x && !(x & (x - 1));
}

// from https://stackoverflow.com/a/12506181
int nextPowerOfTwo(int x) {
	int power = 1;
	while (power < x) {
		power *= 2;
	}
	return power;
}


// from https://stackoverflow.com/a/36095407
// Get the current time in nanoseconds
long get_nanos() {
	struct timespec ts;
	timespec_get(&ts, TIME_UTC);
	return (long)ts.tv_sec * 1000000000L + ts.tv_nsec;
}


/*
	Timings

	'level 1' = blockscan
	'l1 bcao' = blockscan with bcao

	The number before the time is the final element of the scanned array

	20000000 Elements
	  host     : 89997032 in 42338 ms
	  gpu      : 89997032 in 16.285631 ms
	  gpu bcao : 89997032 in 8.554880 ms

	10000000 Elements
	  host     : 44983528 in 20749 ms
	  gpu      : 44983528 in 7.860768 ms
	  gpu bcao : 44983528 in 4.304064 ms

	1000000 Elements
	  host     : 4494474 in 2105 ms
	  gpu      : 4494474 in 0.975648 ms
	  gpu bcao : 4494474 in 0.600416 ms

	10000 Elements
	  host     : 45078 in 19 ms
	  gpu      : 45078 in 0.213760 ms
	  gpu bcao : 45078 in 0.192128 ms

	5000 Elements
	  host     : 22489 in 11 ms
	  gpu      : 22489 in 0.169312 ms
	  gpu bcao : 22489 in 0.148832 ms

	4096 Elements
	  host     : 18294 in 9 ms
	  gpu      : 18294 in 0.132672 ms
	  gpu bcao : 18294 in 0.128480 ms

	2048 Elements
	  host     : 9149 in 4 ms
	  gpu      : 9149 in 0.140736 ms
	  gpu bcao : 9149 in 0.126944 ms

	2000 Elements
	  host     : 8958 in 3 ms
	  gpu      : 8958 in 0.178912 ms
	  gpu bcao : 8958 in 0.214464 ms

	1000 Elements
	  host     : 4483 in 2 ms
	  gpu      : 4483 in 0.020128 ms
	  gpu bcao : 4483 in 0.010784 ms
	  level 1  : 4483 in 0.018080 ms
	  l1 bcao  : 4483 in 0.010400 ms

	500 Elements
	  host     : 2203 in 4 ms
	  gpu      : 2203 in 0.013440 ms
	  gpu bcao : 2203 in 0.009664 ms
	  level 1  : 2203 in 0.013280 ms
	  l1 bcao  : 2203 in 0.010176 ms

	100 Elements
	  host     : 356 in 0 ms
	  gpu      : 356 in 0.008512 ms
	  gpu bcao : 356 in 0.009280 ms
	  level 1  : 356 in 0.008896 ms
	  l1 bcao  : 356 in 0.009056 ms

	64 Elements
	  host     : 221 in 0 ms
	  gpu      : 221 in 0.007584 ms
	  gpu bcao : 221 in 0.008960 ms
	  level 1  : 221 in 0.007360 ms
	  l1 bcao  : 221 in 0.008352 ms

	8 Elements
	  host     : 24 in 0 ms
	  gpu      : 24 in 0.006240 ms
	  gpu bcao : 24 in 0.007392 ms
	  level 1  : 24 in 0.006176 ms
	  l1 bcao  : 24 in 0.007424 ms

	5 Elements
	  host     : 12 in 0 ms
	  gpu      : 12 in 0.006144 ms
	  gpu bcao : 12 in 0.007296 ms
	  level 1  : 12 in 0.006048 ms
	  l1 bcao  : 12 in 0.007328 ms
*/