#include "hip/hip_runtime.h"
#ifndef __GPUSORT
#define __GPUSORT


void merge_serial(int *output, int *index, int *input, int lo, int mid, int hi) 
{
    int i = lo, j = mid + 1;

    for (int k = lo; k <= hi; k++) {
        output[k] = index[k];
    }

    for (int k = lo; k <= hi; k++) {
        if (i > mid) {
            index[k] = output[j++];
        } else if (j > hi) {
            index[k] = output[i++];
        } else if (input[output[i]] <= input[output[j]]) {
            index[k] = output[i++];
        } else {
            index[k] = output[j++];
        }
    }
}

void mergesort_serial(int *output, int *index, int *input, int n, int chunk){
    int chunk_id;
    for(chunk_id=0; chunk_id*chunk<=n; chunk_id++){
        int start = chunk_id * chunk, end, mid;
        if(start >= n) return;
        mid = min(start + chunk/2, n);
        end = min(start + chunk, n);
        merge_serial(output, index, input, start, mid, end);
    }
}


void mergeSort(int *output, int *index, int *input, int lo, int hi) 
{
    if (hi <= lo)
        return;
    int mid = (hi + lo) / 2;
    mergeSort(output, index, input, lo, mid);
    mergeSort(output, index, input, mid + 1, hi);
    merge(output, index, input, lo, mid, hi);
}

void cpuMergeSort(int *output, int *index, int *input, int length) 
{
    mergeSort(output, index, input, 0, length-1);
    for (int i=0; i<length; i++)
        output[i] = input[index[i]];
}

/// Sequential Merge Sort for GPU when Number of Threads Required gets below 1 Warp Size
void mergesort_gpu_seq(int *list, int *sorted, int n, int chunk){
    int chunk_id;
    for(chunk_id=0; chunk_id*chunk<=n; chunk_id++){
        int start = chunk_id * chunk, end, mid;
        if(start >= n) return;
        mid = min(start + chunk/2, n);
        end = min(start + chunk, n);
        merge(list, sorted, start, mid, end);
    }
}


__device__ void merge_gpu(int *a, int *temp, int *perm, int lo, int mid, int hi) 
{
    int i = lo, j = mid + 1;

    for (int k = lo; k <= hi; k++) {
        temp[k] = perm[k];
    }

    for (int k = lo; k <= hi; k++) {
        if (i > mid) {
            perm[k] = temp[j++];
        } else if (j > hi) {
            perm[k] = temp[i++];
        } else if (a[temp[i]] <= a[temp[j]]) {
            perm[k] = temp[i++];
        } else {
            perm[k] = temp[j++];
        }
    }
}

__device__ void merge_gpu(int *list, int *sorted, int start, int mid, int end)
{
    int k=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) sorted[k] = list[i++];
        else if (i==mid) sorted[k] = list[j++];
        else if (list[i]<list[j]) sorted[k] = list[i++];
        else sorted[k] = list[j++];
        k++;
    }
}

__global__ void mergesort_gpu(int *list, int *sorted, int n, int chunk){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunk;
    if(start >= n) return;
    int mid, end;

    mid = min(start + chunk/2, n);
    end = min(start + chunk, n);
    merge_gpu(list, sorted, start, mid, end);
}

// Sequential Merge Sort for GPU when Number of Threads Required gets below 1 Warp Size
void mergesort_gpu_seq(int *list, int *sorted, int n, int chunk){
    int chunk_id;
    for(chunk_id=0; chunk_id*chunk<=n; chunk_id++){
        int start = chunk_id * chunk, end, mid;
        if(start >= n) return;
        mid = min(start + chunk/2, n);
        end = min(start + chunk, n);
        merge(list, sorted, start, mid, end);
    }
}


int mergesort(int *list, int *sorted, int n){

    int *list_d;
    int *sorted_d;
    int dummy;
    bool flag = false;
    bool sequential = false;

    int size = n * sizeof(int);

    hipMalloc((void **)&list_d, size);
    hipMalloc((void **)&sorted_d, size);

    hipMemcpy(list_d, list, size, hipMemcpyHostToDevice);
    hipError_t err = hipGetLastError();
    if(err!=hipSuccess){
        printf("Error_2: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int major = prop.major;
    int minor = prop.minor;
    if(major!=3 || minor!=5){
        printf("The Program is Optimized only for sm_35 Compute Capability..May NOT Work for Other CCs\n");
    }
    // vaues for sm_35 compute capability
    const int max_active_blocks_per_sm = 16;
    const int max_active_warps_per_sm = 64;

    int warp_size = prop.warpSize;
    int max_grid_size = prop.maxGridSize[0];
    int max_threads_per_block = prop.maxThreadsPerBlock;
    int max_procs_count = prop.multiProcessorCount;

    int max_active_blocks = max_active_blocks_per_sm * max_procs_count;
    int max_active_warps = max_active_warps_per_sm * max_procs_count;

    int chunk_size;
    for(chunk_size=2; chunk_size<2*n; chunk_size*=2){
        int blocks_required=0, threads_per_block=0;
        int threads_required = (n%chunk_size==0) ? n/chunk_size : n/chunk_size+1;

        if (threads_required<=warp_size*3 && !sequential){
            sequential = true;
            if(flag) hipMemcpy(list, sorted_d, size, hipMemcpyDeviceToHost);
            else hipMemcpy(list, list_d, size, hipMemcpyDeviceToHost);
            err = hipGetLastError();
            if(err!=hipSuccess){
                printf("ERROR_4: %s\n", hipGetErrorString(err));
                return -1;
            }
            hipFree(list_d);
            hipFree(sorted_d);
        }
        else if (threads_required<max_threads_per_block){
            threads_per_block = warp_size*4;
            dummy = threads_required/threads_per_block;
            blocks_required = (threads_required%threads_per_block==0) ? dummy : dummy+1;
        }
        else if(threads_required<max_active_blocks*warp_size*4){
            threads_per_block = max_threads_per_block/2;
            dummy = threads_required/threads_per_block;
            blocks_required = (threads_required%threads_per_block==0) ? dummy : dummy+1;
        }else{
            dummy = threads_required/max_active_blocks;
            // int estimated_threads_per_block = (dummy%warp_size==0) ? dummy : (dummy/warp_size + 1)*warp_size;
            int estimated_threads_per_block = (threads_required%max_active_blocks==0) ? dummy : dummy+1;
            if(estimated_threads_per_block > max_threads_per_block){
                threads_per_block = max_threads_per_block;
                dummy = threads_required/max_threads_per_block;
                blocks_required = (threads_required%max_threads_per_block==0) ? dummy : dummy+1;
            } else{
                threads_per_block = estimated_threads_per_block;
                blocks_required = max_active_blocks;
            }
        }

        if(blocks_required>=max_grid_size){
            printf("ERROR_2: Too many Blocks Required\n");
            return -1;
        }

        if(sequential){
            // struct timespec start, stop;
            // clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
            mergesort_gpu_seq(list, sorted, n, chunk_size);
            // clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
            // double result = (stop.tv_sec - start.tv_sec) * 1e3 + (stop.tv_nsec - start.tv_nsec) / 1e6;
            // printf("CHUNK SIZE:%d, ", chunk_size);
            // printf("TOTAL THREADS REQUIRED:%d\n", threads_required);
            // printf("TIME TAKEN: %fms\n", result);
            // printf("####################################################\n");
        }else{
            // float time;
            // hipEvent_t start, stop;
            // hipEventCreate(&start);
            // hipEventCreate(&stop);
            // hipEventRecord(start, 0);
            if(flag) mergesort_gpu<<<blocks_required, threads_per_block>>>(sorted_d, list_d, n, chunk_size);
            else mergesort_gpu<<<blocks_required, threads_per_block>>>(list_d, sorted_d, n, chunk_size);
            hipDeviceSynchronize();
            // hipEventRecord(stop, 0);
            // hipEventSynchronize(stop);
            // hipEventElapsedTime(&time, start, stop);
            //
            // printf("CHUNK SIZE:%d, ", chunk_size);
            // printf("TOTAL THREADS REQUIRED:%d, ", threads_required);
            // printf("THREADS PER BLOCK:%d, ", threads_per_block);
            // printf("BLOCKS REQUIRED:%d ", blocks_required);
            // printf("TIME TAKEN: %fms\n", time);
            // printf("####################################################\n");
            err = hipGetLastError();
            if(err!=hipSuccess){
                printf("ERROR_3: %s\n", hipGetErrorString(err));
                return -1;
            }
            flag = !flag;
        }
    }
    return 0;
}

#endif


